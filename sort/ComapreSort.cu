#include "hip/hip_runtime.h"
﻿#include <iostream>
using namespace std;

#include <hip/hip_runtime.h>       
#include <stdlib.h>
#include <stdio.h>
#include <time.h>


#define THREADS pow(2,8)// 256
#define BLOCKS pow(2,13)
#define NUM_VALS THREADS*BLOCKS     //정렬할 원소 개수

int* dynamic_alloc(int n)
{
    int* arr = new int[n];
    return arr;
}

void delete_alloc(int** arr, int n)
{
    for (int j = 0; j < n; j++)
    {
        delete[] arr[j];
    }
    delete[] arr;
}

void print_rt(clock_t start, clock_t stop)
{
    double time = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Run time: %.3fs\n", time);
}


__global__ void bitonic_sort_step(int* dev_values, int j, int k)
{
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;// thread 인덱싱

    ixj = i ^ j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj) > i) {
        if ((i & k) == 0) {
            /* Sort ascending */
            if (dev_values[i] > dev_values[ixj]) {
                /* exchange(i,ixj); */
                int temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i & k) != 0) {
            /* Sort descending */
            if (dev_values[i] < dev_values[ixj]) {
                /* exchange(i,ixj); */
                int temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

void bitonic_sort(int* values)
{
    int* dev_values;
    size_t size = NUM_VALS * sizeof(int);

    hipMalloc((void**)&dev_values, size);                      //쿠다 메모리 할당
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);//정렬할 배열을 dev_values에 복사

    // Block과 Thread는 1차원
    dim3 blocks(BLOCKS, 1);    /* Number of blocks   */
    dim3 threads(THREADS, 1);  /* Number of threads  */

    int j, k;
    /* Major step */
    for (k = 2; k <= NUM_VALS; k <<= 1) {
        /* Minor step */
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonic_sort_step << <blocks, threads >> > (dev_values, j, k);  //dev_values 배열 정렬
            //1 block당 threads가 할당됨 즉 전체 thread 개수는 blocks*threads
        }
    }
    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);           //정렬 완료된 배열을 values에 저장
    hipFree(dev_values);                   //메모리 해제
}

void _swap(int arr[], int a, int b)
{
    int temp = arr[b];
    arr[b] = arr[a];
    arr[a] = temp;
}

void InsertSort(int a[], int l, int r)
{
    int i, j, temp;
    for (i = l + 1; i <= r; i++)
    {
        temp = a[i];
        for (j = i - 1; j >= l && a[j] > temp; j--)
            a[j + 1] = a[j];
        a[j + 1] = temp;
    }
}

//Quick Sort - a : 배열 / l : 시작 인덱스 / r : 끝 인덱스
void ThreeMedianInsertQS(int a[], int l, int r, int n)
{
    int i, j, m, v;
    if (r - l <= n)     //배열의 크기가 n보다 작거나 같을시, 삽입정렬 수행
        InsertSort(a, l, r);
    else if (r - l > 1) //배열의 크기가 2보다 크면
    {
        m = (l + r) / 2; //중간 인덱스값 m

        //중간값을 이용하여 성능 향상
        //a[l], a[m], a[r] 값중 중간값이 m 인덱스에 가도록 swap
        if (a[l] > a[m])
            _swap(a, l, m);
        if (a[l] > a[r])
            _swap(a, l, r);
        if (a[m] > a[r])
            _swap(a, m, r);

        _swap(a, m, r - 1); //중간값 a[m]을 가장 끝 인덱스 r 로 이동
        v = a[r - 1];       //피벗 값
        i = l;              //left 포인터
        j = r - 1;          //right 포인터

        //피벗값을 기준으로 좌측 부분(피벗보다 작은 값), 우측 부분(피벗보다 큰 값)으로 분할
        for (;;)
        {
            while (a[++i] < v)
                ; //left 포인터가 가리키는 값이 피벗보다 작으면 포인터 우측 이동
            while (a[--j] > v)
                ; //right 포인터가 가리키는 값이 피벗보다 크면 포인터 좌측 이동
            if (i >= j)
                break; //left 포인터와 right 포인터가 교차하면, 반복문 탈출

            //left 포인터가 가리키는 값이 피벗보다 크거나, right 포인터가 가리키는 값이 피벗보다 작으면
            //if(a[i] > v || a[j] < v)
            _swap(a, i, j); //두 값 swap
        }
        _swap(a, i, r - 1);     //피벗 값을 좌측 부분과 우측 부분 사이로 이동
        ThreeMedianInsertQS(a, l, i - 1, n); //좌측 부분 재귀 호출
        ThreeMedianInsertQS(a, i + 1, r, n); //우측 부분 재귀 호출
    }
    else if (a[l] > a[r]) //배열의 크기가 2보다 작을 때, 좌측 값이 우측 값보다 크면
        _swap(a, l, r);   //두 값 swap
}

// v : 삽입 정렬 key값 / k : gap sequence index
void ShellSort(int a[], int n)
{
    int i, j, v, k;
    int gap[20];

    gap[0] = 1;     //gap sequence 초항은 1
    k = 0;

    while (gap[k] < n) {
        k++;
        gap[k] = pow(4, k) + 3 * pow(2, k - 1) + 1;//Sedgewick gap sequence
    }

    for (; k >= 0; k--) {
        for (i = gap[k]; i < n; i++) {      //gap만큼 점프하며 삽입 정렬
            v = a[i];
            j = i;
            while (j >= gap[k] && a[j - gap[k]] > v) {
                a[j] = a[j - gap[k]];
                j = j - gap[k];
            }
            a[j] = v;
        }
    }
}


// sorting이 정순으로 되었는지 check
void checkSort(int a[], int n)
{
    int i, sorted;
    sorted = true;
    for (i = 0; i < n - 1; i++)
    {
        if (a[i] > a[i + 1])
        {
            sorted = false;
        }
        if (!sorted)
        {
            break;
        }
    }
    if (sorted)
    {
        cout << "Sorting complete!" << endl;
    }
    else
    {
        cout << "Error during sorting..." << endl;
    }
}


int main(void) {
    clock_t start, stop;
    int* a1 = dynamic_alloc(NUM_VALS);
    int* a2 = dynamic_alloc(NUM_VALS);
    int* a3 = dynamic_alloc(NUM_VALS);

    cout << "Number of Values : " << NUM_VALS << "\n";

    srand(time(NULL));
    for (int i = 0; i < NUM_VALS; i++)
    {
        a1[i] = rand() % RAND_MAX;
        a2[i] = a1[i];
        a3[i] = a1[i];
    }
    start = clock();
    bitonic_sort(a1);
    stop = clock();
    checkSort(a1, NUM_VALS);
    cout << "Bitonic_cuda ";
    print_rt(start, stop);

    start = clock();
    ThreeMedianInsertQS(a2, 0, NUM_VALS - 1, 15);
    stop = clock();
    checkSort(a2, NUM_VALS);
    cout << "Quick ";
    print_rt(start, stop);

    start = clock();
    ShellSort(a3, NUM_VALS);
    stop = clock();
    checkSort(a3, NUM_VALS);
    cout << "Shell ";
    print_rt(start, stop);

}